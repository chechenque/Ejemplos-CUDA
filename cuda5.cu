
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define TPB 8
#define W 4
#define H 4
#define TX 1
#define TY 1

int N=H*W;


__device__ float distance(float x1, float x2){
 return sqrt ((x2-x1)*(x2-x1));
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref, int w){
	const int c=blockIdx.x*blockDim.x+threadIdx.x;
	const int r=blockIdx.y*blockDim.y+threadIdx.y;
	const int i=r*w+c;
	
	const float x=d_in[i];
	d_out[i]=distance(x, ref);

	printf("c=%d, r=%d, i=%d: the distance between %f to %f is %f. \n", c, r, i, ref, x, d_out[i]); ////
}

void distanceArray(float *out, float *in, float ref, int len){
	float *d_in=0;
	float *d_out=0; ////

	hipMalloc(&d_in, len*sizeof(float));
	hipMalloc(&d_out, len*sizeof(float));

	hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);

	const dim3 blockSize(TX, TY);
	const int bx=(W+TX-1)/TX;
	const int by=(W+TY-1)/TY;
	const dim3 gridSize=dim3(bx, by);

	distanceKernel<<<gridSize, blockSize>>>(d_out, d_in, ref, W);

	hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
}

float scale(int i, int n){
 return ((float) i)/(n-1);
}

int main(){
	const float ref=0.5f;
	
	float *in=(float*) calloc(N,sizeof(float));
	float *out=(float*) calloc(N, sizeof(float));

	for(int i=0; i<N; ++i){
		in[i]=scale(i,N); //
	}

	distanceArray(out, in, ref, N);

	printf("______________________________ \n");

	for(int j=0; j<N; ++j){
		printf("The distance, printed from the host, between %f to %f is %f. \n", ref, in[j], out[j]);
	}

	free(in);
	free(out);

	return 0;
}